
#include <hip/hip_runtime.h>
#include <iostream>
#include <list>
#include <fstream>
#include <set>
#include <map>
#include <vector>
#include <iterator>
#include <list> 
#include <queue>
#include <chrono>
#include <stdlib.h>

using namespace std;
using namespace std::chrono;

//Number of vertices
int V = 15; 	//8297;  //73; 
//Number of Edges
int E = 28; 	//103689;  //100; 

// This class represents a directed graph using compressed sparse row representation
class Graph
{
	public:

	int V;    // No. of vertices
	int E;    // No. of Edges
	int maxColour;
	 
	int* edges;
	int* nodes;
	int* colour;
	int* marked;


	Graph(char* filename, int V, int E);  // Constructor
	~Graph()    //Destructor
	{
		free(edges);
		free(nodes);
		free(marked);
		free(colour);
	}
	
	void buildCSRGraph(char filename[]); //Create Graph
	__device__ void printInfo();
	__device__ int checkIndegree(int);
	__device__ int checkOutdegree(int);
	__device__ 	int isEdge(int, int);

	
};

Graph::Graph(char filename[], int V, int E)
{
	this->V = V;
	this->E = E;
	this->maxColour = 0;
	this->nodes = (int *)malloc(sizeof(int)*V);
	this->colour = (int *)calloc(V, sizeof(int));
	this->marked = (int *)calloc(V, sizeof(int));
	this->edges = (int *)malloc(sizeof(int)*E);
	
	int i;
	for (i = 0; i<V; i++)
	{
		this->nodes[i] = -1;
	}
	buildCSRGraph(filename);
	
}

void Graph::buildCSRGraph(char filename[])
{
	printf("Building CSR...\n");
	int count = 0;
	unsigned int s, d;
	std::ifstream infile(filename);
	
	if (!infile.is_open())
	{
		printf("Could not open Data file\n");
		return;
	}

	int prev = -1;
	int count_edges = 0;

	while (infile >> s >> d)
	{
		count_edges++;
		//printf("%d, %d\n",s, d);
		if (prev == -1 || s != prev)
		{
			nodes[s] = count;
		}
		edges[count] = d;
		count++;
		prev = s;
	}

	infile.close();

	printf("\nDone building CSR!\n");
}

__device__ void Graph::printInfo()
{
	int i;

	//To print nodes and edges arrays
	for (i = 0; i<V; i++)
		printf("\nNodes[%d] = %d", i, nodes[i]);
	printf("\nEdges:");
	for (i = 0; i<E; i++)
		printf("%d ", edges[i]);
	printf("\n");
}

__device__ int Graph::checkIndegree(int i)
{
	if (marked[i]) return -1;
	int j;
	int validInEdges = 0;
	for (j = 0; j<E; j++)
	{
		if (edges[j] == i)
		{
			int min = -1;
			int ind = -1, k;
			for (k = 0; k<V; k++)
			if (nodes[k] > min && nodes[k] <= j){
				min = nodes[k];
				ind = k;
			}
			if (marked[ind]) continue;
			validInEdges++;
		}
	}
	return validInEdges;
}

__device__ int Graph::checkOutdegree(int i)
{
	if (marked[i]) return -1;
	if (nodes[i] == -1) return 0;
	int k, h;
	k = i + 1;
	int end;
	while (k<V)
	{
		if (nodes[k] != -1)
			break;
		k++;
	}
	int validOutEdges = 0;
	end = nodes[k];
	if (k == V) end = E;
	for (h = nodes[i]; h<end; h++)
	{
		if (marked[edges[h]] == 0)
			validOutEdges++;
	}
	return validOutEdges;
}


__global__ void Trim1(Graph* d_g)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if(i >= d_g->V )
	   return;

	//printf("\n Processing Node %d", i);
	if (d_g->checkOutdegree(i) == 0)
	{
		//printf("\nOutdegree is zero for %d\n",i);
		d_g->marked[i] = 1;
		d_g->maxColour++;
		d_g->colour[i] = d_g->maxColour;
		return;
	}
	else if (d_g->checkIndegree(i) == 0)
	{
		//printf("\nIndegree is zero for %d\n",i);
		d_g->marked[i] = 1;
		d_g->maxColour++;
		d_g->colour[i] = d_g->maxColour;
	}
   
}

__device__ int Graph::isEdge(int i, int j)
{
	if (i == j) return 0;
	int k, h;
	if (nodes[i] == -1) return 0;
	k = i + 1;
	while (k<V)
	{
		if (nodes[k] != -1)
			break;
		k++;
	}
	for (h = nodes[i]; h<nodes[k]; h++)
	{
		if (edges[h] == j)
			return 1;
	}
	return 0;
}

__global__ void Trim2(Graph* d_g)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if(i >= d_g->V )
	   return;

    int j;
	for (j = 0; j<d_g->V; j++)
	{
		if (d_g->marked[i] || d_g->marked[j]) continue;
		if (d_g->isEdge(i, j) && d_g->isEdge(j, i))
		{

			if ((d_g->checkIndegree(i) == 1 && d_g->checkIndegree(j) == 1) || (d_g->checkOutdegree(i) == 1 && d_g->checkOutdegree(j) == 1))
			{
				d_g->marked[i] = d_g->marked[j] = 1;
				d_g->maxColour++;
				d_g->colour[i] = d_g->colour[j] = d_g->maxColour;
			}
		}
	}
}

void ColourMapFunction(int *colours, int mc)
{
	int i;
	map <int, std::vector<int> > colourMap;

    printf("\nBuilding the Colour Hash Map");
	for (i = 0; i < mc; i++)
	{
		vector<int> nodeList;
		colourMap.insert(pair <int, std::vector<int> >(i, nodeList));
	}
	for (i = 0; i < V; i++)
	{
		colourMap[colours[i]].push_back(i);
	}

    
	map <int, std::vector<int> > ::iterator itr;
	vector<int>::iterator jtr;
	printf("\nThe Colour Hash Map is as follows \n");
	for (itr = colourMap.begin(); itr != colourMap.end(); ++itr)
	{
		cout << '\t' << itr->first << '\t';
		for (jtr = itr->second.begin(); jtr != itr->second.end(); jtr++)
		{
			cout << *jtr << " ";
		}
		cout << "\n";
	}

    printf("\nFinding the biggest SCC in the graph");
    int max_size = -1, size;
	for (itr = colourMap.begin(); itr != colourMap.end(); ++itr)
	{
		size = itr->second.size();
		if(max_size < size)
		   max_size = size;		
	}
	printf("\nSize of the biggest SCC is %d ", max_size);

}

void SCC(Graph* d_g)
{
	int threads, blocks ;
	
	threads = 1024;
	blocks = V/threads + 1;
	
	//Trim 1
	printf("\nTrim1 ....");
	Trim1<<<blocks, threads >>>(d_g);
	hipDeviceSynchronize();
	printf("\nDone with Trim1 ....");

    //buildColourMap<<<1,1>>>(d_g);
    //FWBW

	/*//Trim 1
	printf("\nTrim1 ....");
	Trim1<<<blocks, threads >>>(d_g);
	cudaDeviceSynchronize();
	printf("\nDone with Trim1 ....");

	//Trim 2
	printf("\nTrim2 ....");
	Trim2<<<blocks, threads >>>(d_g);
	cudaDeviceSynchronize();
	printf("\nDone with Trim2 ....");

	//Trim 1
	printf("\nTrim1 ....");
	Trim1<<<blocks, threads >>>(d_g);
	cudaDeviceSynchronize();
	printf("\nDone with Trim1 ....");*/

	
	//WCC

	//repeated_FWBW();

	return;
}

int main(int argc, char* argv[])
{
	//Data Filename
	char filename[] = "./smallDummyDataSorted.txt";

	Graph h_g(filename, V, E);
	Graph *d_g;

	//Copy data from host to device
	hipMalloc((void **)&d_g, sizeof(Graph));
	hipMemcpy(d_g, &h_g, sizeof(Graph), hipMemcpyHostToDevice);

	int *h_edges, *h_nodes, *h_marked, *h_colour;
    hipMalloc((void **)&h_edges, sizeof(int)*E);
	hipMemcpy(h_edges, h_g.edges, sizeof(int)*E, hipMemcpyHostToDevice);
	hipMalloc((void **)&h_nodes, sizeof(int)*V);
	hipMemcpy(h_nodes, h_g.nodes, sizeof(int)*V, hipMemcpyHostToDevice);
	hipMalloc((void **)&h_marked, sizeof(int)*V);
	hipMemcpy(h_marked, h_g.marked, sizeof(int)*V, hipMemcpyHostToDevice);
	hipMalloc((void **)&h_colour, sizeof(int)*V);
    hipMemcpy(h_colour, h_g.colour, sizeof(int)*V, hipMemcpyHostToDevice);
    
	hipMemcpy(&(d_g->edges), &h_edges, sizeof(int *), hipMemcpyHostToDevice);
	hipMemcpy(&(d_g->nodes), &h_nodes, sizeof(int *), hipMemcpyHostToDevice);
	hipMemcpy(&(d_g->marked), &h_marked, sizeof(int *), hipMemcpyHostToDevice);
	hipMemcpy(&(d_g->colour), &h_colour, sizeof(int *), hipMemcpyHostToDevice);

    //Find SCCs in the graph
	SCC(d_g);
  
	//Copy data from device to host
	int *h_colour_ret, *h_nodes_ret, *h_marked_ret, *h_edges_ret;
	h_colour_ret = (int*)malloc(sizeof(int)*V);
	h_nodes_ret = (int*)malloc(sizeof(int)*V);
	h_marked_ret = (int*)malloc(sizeof(int)*V);
	h_edges_ret = (int*)malloc(sizeof(int)*E);
	hipMemcpy(&h_g, d_g,sizeof(Graph),hipMemcpyDeviceToHost); 
	//cudaMemcpy(&(h_g.colour), &(d_g->colour), sizeof(int*), cudaMemcpyDeviceToHost);
	hipMemcpy(h_colour_ret, h_g.colour, sizeof(int)*V, hipMemcpyDeviceToHost );
	hipMemcpy(h_marked_ret, h_g.marked, sizeof(int)*V, hipMemcpyDeviceToHost );
	hipMemcpy(h_nodes_ret, h_g.nodes, sizeof(int)*V, hipMemcpyDeviceToHost );
	hipMemcpy(h_edges_ret, h_g.edges, sizeof(int)*E, hipMemcpyDeviceToHost );
	
	//Build colour map, print it and find the size of the largest SCC
	ColourMapFunction(h_colour_ret, h_g.maxColour);

	hipFree(d_g);
	hipFree(h_edges);
	hipFree(h_nodes);
	hipFree(h_marked);
	hipFree(h_colour);

	free(h_colour_ret);
	free(h_marked_ret);
	free(h_nodes_ret);
	free(h_edges_ret);
	hipDeviceReset();

	printf("\n");
	return 0;
}
