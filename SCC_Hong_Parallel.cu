
#include <hip/hip_runtime.h>
#include <iostream>
#include <list>
#include <fstream>
#include <set>
#include <map>
#include <vector>
#include <iterator>
#include <list> 
#include <queue>
#include <chrono>
#include <stdlib.h>

using namespace std;
using namespace std::chrono;

//Number of vertices
int V = 15; 	//8297;  //73; 
//Number of Edges
int E = 28; 	//103689;  //100; 

// This class represents a directed graph using compressed sparse row representation
class Graph
{
	public:

	int V;    // No. of vertices
	int E;    // No. of Edges
	int maxColour;
	 
	int* edges;
	int* nodes;
	int* colour;
	int* marked;
	
	Graph(char* filename, int V, int E);  // Constructor
	~Graph()    //Destructor
	{
		free(edges);
		free(nodes);
		free(marked);
		free(colour);
	}
	
	void buildCSRGraph(char filename[]); //Create Graph
	__device__ void printInfo();
	__device__ int checkIndegree(int);
	__device__ int checkOutdegree(int);
	__device__ 	int isEdge(int, int);

	
};

Graph::Graph(char filename[], int V, int E)
{
	this->V = V;
	this->E = E;
	this->maxColour = 0;
	this->nodes = (int *)malloc(sizeof(int)*V);
	this->colour = (int *)calloc(V, sizeof(int));
	this->marked = (int *)calloc(V, sizeof(int));
	this->edges = (int *)malloc(sizeof(int)*E);
	
	int i;
	for (i = 0; i<V; i++)
	{
		this->nodes[i] = -1;
	}
	buildCSRGraph(filename);
	
}

void Graph::buildCSRGraph(char filename[])
{
	printf("Building CSR...\n");
	int count = 0;
	unsigned int s, d;
	std::ifstream infile(filename);
	
	if (!infile.is_open())
	{
		printf("Could not open Data file\n");
		return;
	}

	int prev = -1;
	int count_edges = 0;

	while (infile >> s >> d)
	{
		count_edges++;
		//printf("%d, %d\n",s, d);
		if (prev == -1 || s != prev)
		{
			nodes[s] = count;
		}
		edges[count] = d;
		count++;
		prev = s;
	}

	infile.close();

	printf("\nDone building CSR!\n");
}

__device__ void Graph::printInfo()
{
	int i;

	//To print nodes and edges arrays
	for (i = 0; i<V; i++)
		printf("\nNodes[%d] = %d", i, nodes[i]);
	printf("\nEdges:");
	for (i = 0; i<E; i++)
		printf("%d ", edges[i]);
	printf("\n");
}

__device__ int Graph::checkIndegree(int i)
{
	if (marked[i]) return -1;
	int j;
	int validInEdges = 0;
	for (j = 0; j<E; j++)
	{
		if (edges[j] == i)
		{
			int min = -1;
			int ind = -1, k;
			for (k = 0; k<V; k++)
			if (nodes[k] > min && nodes[k] <= j){
				min = nodes[k];
				ind = k;
			}
			if (marked[ind]) continue;
			validInEdges++;
		}
	}
	return validInEdges;
}

__device__ int Graph::checkOutdegree(int i)
{
	if (marked[i]) return -1;
	if (nodes[i] == -1) return 0;
	int k, h;
	k = i + 1;
	int end;
	while (k<V)
	{
		if (nodes[k] != -1)
			break;
		k++;
	}
	int validOutEdges = 0;
	end = nodes[k];
	if (k == V) end = E;
	for (h = nodes[i]; h<end; h++)
	{
		if (marked[edges[h]] == 0)
			validOutEdges++;
	}
	return validOutEdges;
}


__global__ void Trim1(Graph* d_g)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if(i >= d_g->V )
	   return;

	//printf("\n Processing Node %d", i);
	if (d_g->checkOutdegree(i) == 0)
	{
		//printf("\nOutdegree is zero for %d\n",i);
		d_g->marked[i] = 1;
		d_g->maxColour++;
		d_g->colour[i] = d_g->maxColour;
		return;
	}
	else if (d_g->checkIndegree(i) == 0)
	{
		//printf("\nIndegree is zero for %d\n",i);
		d_g->marked[i] = 1;
		d_g->maxColour++;
		d_g->colour[i] = d_g->maxColour;
	}
   
}

__device__ int Graph::isEdge(int i, int j)
{
	if (i == j) return 0;
	int k, h;
	if (nodes[i] == -1) return 0;
	k = i + 1;
	while (k<V)
	{
		if (nodes[k] != -1)
			break;
		k++;
	}
	for (h = nodes[i]; h<nodes[k]; h++)
	{
		if (edges[h] == j)
			return 1;
	}
	return 0;
}

__global__ void Trim2(Graph* d_g)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if(i >= d_g->V )
	   return;

    int j;
	for (j = 0; j<d_g->V; j++)
	{
		if (d_g->marked[i] || d_g->marked[j]) continue;
		if (d_g->isEdge(i, j) && d_g->isEdge(j, i))
		{

			if ((d_g->checkIndegree(i) == 1 && d_g->checkIndegree(j) == 1) || (d_g->checkOutdegree(i) == 1 && d_g->checkOutdegree(j) == 1))
			{
				d_g->marked[i] = d_g->marked[j] = 1;
				d_g->maxColour++;
				d_g->colour[i] = d_g->colour[j] = d_g->maxColour;
			}
		}
	}
}

void SCC(Graph* d_g)
{
	int threads, blocks ;
	
	//Trim 1
	threads = 1024;
	blocks = V/threads + 1;
	
	printf("\nTrim1 ....");
	Trim1<<<blocks, threads >>>(d_g);
	hipDeviceSynchronize();
	printf("\nDone with Trim1 ....");

	//Trim 2
	printf("\nTrim2 ....");
	Trim2<<<blocks, threads >>>(d_g);
	hipDeviceSynchronize();
	printf("\nDone with Trim2 ....");


}

int main(int argc, char* argv[])
{
	//Data Filename
	char filename[] = "./smallDummyDataSorted.txt";

	Graph h_g(filename, V, E);

	Graph *d_g;
	hipMalloc((void **)&d_g, sizeof(Graph));
	hipMemcpy(d_g, &h_g, sizeof(Graph), hipMemcpyHostToDevice);

	int *h_edges, *h_nodes, *h_marked, *h_colour;
    hipMalloc((void **)&h_edges, sizeof(int)*E);
	hipMemcpy(h_edges, h_g.edges, sizeof(int)*E, hipMemcpyHostToDevice);
	hipMalloc((void **)&h_nodes, sizeof(int)*V);
	hipMemcpy(h_nodes, h_g.nodes, sizeof(int)*V, hipMemcpyHostToDevice);
	hipMalloc((void **)&h_marked, sizeof(int)*V);
	hipMemcpy(h_marked, h_g.marked, sizeof(int)*V, hipMemcpyHostToDevice);
	hipMalloc((void **)&h_colour, sizeof(int)*V);
    hipMemcpy(h_colour, h_g.colour, sizeof(int)*V, hipMemcpyHostToDevice);
    
	hipMemcpy(&(d_g->edges), &h_edges, sizeof(int *), hipMemcpyHostToDevice);
	hipMemcpy(&(d_g->nodes), &h_nodes, sizeof(int *), hipMemcpyHostToDevice);
	hipMemcpy(&(d_g->marked), &h_marked, sizeof(int *), hipMemcpyHostToDevice);
	hipMemcpy(&(d_g->colour), &h_colour, sizeof(int *), hipMemcpyHostToDevice);

	SCC(d_g);
	printf("\n");
	return 0;
}
